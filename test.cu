#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>


int main (int argc, char *argv[])
{
	hipError_t r;
	hipDevice_t dev;
	hipDeviceptr_t mem;
	int device_count;
	hipCtx_t ctx;

	printf ("Initializing...\n");
	r = hipInit (0);
	assert (r == hipSuccess);

	r = hipGetDeviceCount (&device_count);
	assert (r == hipSuccess);
	assert (device_count > 0);

	r = hipDeviceGet (&dev, 0);
	assert (r == hipSuccess);

	r = hipCtxCreate (&ctx, 0, dev);
	assert (r == hipSuccess);

	printf ("Init done.\n");


	printf ("Allocating...\n");
	r = hipMalloc (&mem, 1234);
	assert (r == hipSuccess);
	printf ("Allocating done.\n");

	printf ("Freeing...\n");
	r = hipFree (mem);
	assert (r == hipSuccess);
	printf ("Freeing done.\n");
}
